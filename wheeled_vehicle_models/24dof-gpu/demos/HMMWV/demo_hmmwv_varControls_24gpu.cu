// =============================================================================
// Authors: Huzaifa Unjhawala
// =============================================================================
//
// This demo describes simulating user 1000 HMMWVs (specified with JSON files), 500 operating on one driver inpput file 
// and the rest operating on another driver input file. Since the Half Implicit solver is the only one supported 
// for the GPU models, that is what is used here.
// Use ./executable_name <threads_per_block>
//
// =============================================================================
#include <hip/hip_runtime.h>
#include <iostream>
#include <random>
#include <hip/hip_runtime.h>
#include <math.h>
#include <numeric>
#include <algorithm>
#include <iterator>
#include <chrono>

#include "dof24_halfImplicit_gpu.cuh"


using namespace d24GPU;
int main(int argc, char** argv) {
    // Set the total number of vehicles
    unsigned int num_vehicles = 1000;
    // Set the threads per block from command line
    unsigned int threads_per_block = std::stoul(argv[1]);

    // Get two driver files
    std::string file_name_1 = "acc3";
    // Driver inputs -> All vehicles have the same driver inputs
    std::string driver_file_1 = "../../24dof-gpu/data/input/" + file_name_1 + ".txt";

    std::string file_name_2 = "double_lane4";
    // Driver inputs -> All vehicles have the same driver inputs
    std::string driver_file_2 = "../../24dof-gpu/data/input/" + file_name_2 + ".txt";

    // Vehicle specification -> We assume that all vehicles have the same parameters
    std::string vehParamsJSON = (char*)"../../24dof-gpu/data/json/HMMWV/vehicle.json";
    std::string tireParamsJSON = (char*)"../../24dof-gpu/data/json/HMMWV/tmeasy.json";
    std::string susParamsJSON = (char*)"../../24dof-gpu/data/json/HMMWV/suspension.json";

    // Construct the solver
    d24SolverHalfImplicitGPU solver(num_vehicles);

    // First construct half the vehicles for driver file 1
    solver.Construct(vehParamsJSON, tireParamsJSON, susParamsJSON, 500, driver_file_1);
    // Then construct the other half for driver file 2
    solver.Construct(vehParamsJSON, tireParamsJSON, susParamsJSON, 500, driver_file_2);

    // Set the threads per block
    solver.SetThreadsPerBlock(threads_per_block);

    // Set the time step of the solver
    solver.SetTimeStep(1e-3);

    // Now we initialize the states -> These are all set to 0 (struct initializer)
    VehicleState veh_st;
    TMeasyState tirelf_st;
    TMeasyState tirerf_st;
    TMeasyState tirelr_st;
    TMeasyState tirerr_st;
    SuspensionState suslf_st;
    SuspensionState susrf_st;
    SuspensionState suslr_st;
    SuspensionState susrr_st;
    // Again we initialize the same states for all vehicles
    solver.Initialize(veh_st, tirelf_st, tirerf_st, tirelr_st, tirerr_st, suslf_st, susrf_st, suslr_st, susrr_st,
                      num_vehicles);
    // This is the end time of the longer driver file
    // Note: This means that the last control input is applied till the end of the simulation for the shorter driver
    // file
    solver.SetEndTime(22.0);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    solver.Solve();
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "Solve time (ms): " << milliseconds << "\n";

    // Extract terminal state of choosen vehicles and print the position
    SimState sim_state_1 = solver.GetSimState(499);
    SimState sim_state_2 = solver.GetSimState(999);

    std::cout << "X Position of vehicle 499: " << sim_state_1._v_states._x << std::endl;
    std::cout << "X Position of vehicle 999: " << sim_state_2._v_states._x << std::endl;
}
