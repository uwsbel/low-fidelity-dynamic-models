// =============================================================================
// Authors: Huzaifa Unjhawala
// =============================================================================
//
// This demo describes simulating user-provided number of HMMWVs (specified with JSON files) on a step-by-step
// basis using the Half Implicit solver on GPU. It includes functionality for setting control inputs through a functor.
// The API setup requires specifying the number of vehicles and threads per block.
// Usage: ./executable_name <total_number_of_vehicles> <threads_per_block>
//
// =============================================================================
#include <hip/hip_runtime.h>
#include <iostream>
#include <filesystem>
#include <hip/hip_runtime.h>
#include <math.h>
#include <numeric>
#include <algorithm>
#include <iterator>
#include <chrono>

#include "dof24_halfImplicit_gpu.cuh"

namespace fs = std::filesystem;
using namespace d24GPU;

// Functor for control inputs
struct MyFunctor {
    __device__ void operator()(double t, DriverInput* controls) {
        if (t < 1) {
            controls->m_steering = 0.0;
            controls->m_throttle = 0.0;
            controls->m_braking = 0.0;
        } else {
            controls->m_steering = 0.0;
            controls->m_throttle = 0.5;
            controls->m_braking = 0.0;
        }
    }
};

int main(int argc, char** argv) {
    if (argc != 3) {
        std::cerr << "Usage: " << argv[0] << " <total_number_of_vehicles> <threads_per_block>" << std::endl;
        return 1;
    }

    unsigned int num_vehicles = std::stoul(argv[1]);
    unsigned int threads_per_block = std::stoul(argv[2]);

    std::string file_name = "acc3";
    std::string inputPath = "../../24dof-gpu/data/input/" + file_name + ".txt";

    // Verify the existence of the input file
    if (!fs::exists(inputPath)) {
        std::cerr << "Error: Input file does not exist: " << inputPath << std::endl;
        return 1;
    }

    std::string vehParamsJSON = "../../24dof-gpu/data/json/HMMWV/vehicle.json";
    std::string tireParamsJSON = "../../24dof-gpu/data/json/HMMWV/tmeasy.json";
    std::string susParamsJSON = "../../24dof-gpu/data/json/HMMWV/suspension.json";

    // Construct the solver with GPU-specific settings
    d24SolverHalfImplicitGPU solver(num_vehicles);
    solver.Construct(vehParamsJSON, tireParamsJSON, susParamsJSON, num_vehicles, inputPath);

    solver.SetThreadsPerBlock(threads_per_block);
    solver.SetTimeStep(1e-3);
    solver.SetKernelSimTime(0.1);  // Set control input timestep

    // Initialize vehicle states
    VehicleState veh_st;
    TMeasyState tirelf_st;
    TMeasyState tirerf_st;
    TMeasyState tirelr_st;
    TMeasyState tirerr_st;
    SuspensionState suslf_st;
    SuspensionState susrf_st;
    SuspensionState suslr_st;
    SuspensionState susrr_st;
    solver.Initialize(veh_st, tirelf_st, tirerf_st, tirelr_st, tirerr_st, suslf_st, susrf_st, suslr_st, susrr_st,
                      num_vehicles);

    // Use functor for control inputs
    MyFunctor myControlsFunctor;
    double endTime = 10.0;
    double timeStep = solver.GetStep();
    double t = 0;
    double new_time = 0;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    while (t < (endTime - timeStep / 10.)) {
        new_time = solver.SolveStep(t, myControlsFunctor);  // Solve for the current time
        t = new_time;
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "Solve time (ms): " << milliseconds << "\n";

    // Extract and print the position of a vehicle
    SimState sim_state_1 = solver.GetSimState(0);
    std::cout << "X Position of vehicle 1: " << sim_state_1._v_states._x << std::endl;

    return 0;
}
