// =============================================================================
// Authors: Huzaifa Unjhawala
// =============================================================================
//
// This demo describes simulating user provided number of HMMWVs (specified with JSON files), all operating on the 
// same driver inputs on the GPU. Since the Half Implicit solver is the only one supported for the GPU models,
// that is what is used here. The structure of the API is very similar to the CPU version except for the 
// additional requirements of specifying the number of vehicles and threads per block. 
// Use ./executable_name <total_number_of_vehicles> <threads_per_block>
//
// =============================================================================
#include <hip/hip_runtime.h>
#include <iostream>
#include <random>
#include <hip/hip_runtime.h>
#include <math.h>
#include <numeric>
#include <algorithm>
#include <iterator>
#include <chrono>

#include "dof24_halfImplicit_gpu.cuh"



using namespace d24GPU;
int main(int argc, char** argv) {
    // Get total number of vehicles from command line
    unsigned int num_vehicles = std::stoul(argv[1]);
    // Set the threads per block from command line
    unsigned int threads_per_block = std::stoul(argv[2]);

    std::string file_name = "acc3";
    // Driver inputs -> All vehicles have the same driver inputs
    std::string driver_file = "../../24dof-gpu/data/input/" + file_name + ".txt";

    // Vehicle specification -> We assume that all vehicles have the same parameters
    std::string vehParamsJSON = (char*)"../../24dof-gpu/data/json/HMMWV/vehicle.json";
    std::string tireParamsJSON = (char*)"../../24dof-gpu/data/json/HMMWV/tmeasy.json";
    std::string susParamsJSON = (char*)"../../24dof-gpu/data/json/HMMWV/suspension.json";

    // Construct the solver
    d24SolverHalfImplicitGPU solver(num_vehicles);
    // The number of vehicles here sets these parameters and inputs for all these vehicles
    // If there is a need to set different parameters for different vehicles, then the solver
    // needs to be constructed for each vehicle separately (using the same sovler object)
    solver.Construct(vehParamsJSON, tireParamsJSON, susParamsJSON, num_vehicles, driver_file);

    // Set the threads per block
    solver.SetThreadsPerBlock(threads_per_block);

    // Set the time step of the solver
    solver.SetTimeStep(1e-3);

    // Now we initialize the states -> These are all set to 0 (struct initializer)
    VehicleState veh_st;
    TMeasyState tirelf_st;
    TMeasyState tirerf_st;
    TMeasyState tirelr_st;
    TMeasyState tirerr_st;
    SuspensionState suslf_st;
    SuspensionState susrf_st;
    SuspensionState suslr_st;
    SuspensionState susrr_st;
    // Again we initialize the same states for all vehicles
    solver.Initialize(veh_st, tirelf_st, tirerf_st, tirelr_st, tirerr_st, suslf_st, susrf_st, suslr_st, susrr_st,
                      num_vehicles);

    // Enable output for all the vehicles
    // solver.SetOutput("../../24dof-gpu/data/output/" + file_name + "_hmmwv24", 100, true);
    // Enable output for 50 of the vehicles
    solver.SetOutput("../../24dof-gpu/data/output/" + file_name + "_hmmwv24", 100, false, 50);
    // Set the simulation end time -> This is a input that *must* be set by the user
    solver.SetEndTime(10.0);

    // Solve
    // Time the solve
    auto start = std::chrono::high_resolution_clock::now();
    solver.Solve();
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed = end - start;
    std::cout << "Solve time: " << elapsed.count() << " s\n";
}
