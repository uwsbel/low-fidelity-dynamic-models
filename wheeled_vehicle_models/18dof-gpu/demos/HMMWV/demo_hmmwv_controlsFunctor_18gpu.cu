// =============================================================================
// Authors: Huzaifa Unjhawala
// =============================================================================
//
// This demo describes simulating user provided number of HMMWVs (specified with JSON files) on a step-by-step
// basis. The "step" at which a new control input is provided can be set with SetKernelSimTime as shown here.
// Additionally, shown here is a way to provide a function object (functor) to the solver to provide the
// control inputs. This is done by creating a functor and passing it to the solver's SolveStep function.
// The functor needs to have an operator() function that takes in the current time and a pointer to the
// DriverInput struct. This functionallity is useful when the control Functor represents some sort of controller.
// Since the Half Implicit solver is the only one supported for the GPU models, that is what is used here.
// When the solver is used in a step-by-step manner, the output is not stored in a file (unlike the CPU models).
// However, access to the vehicle states every control time step is provided through the GetSimState function.
// Use ./executable_name <total_number_of_vehicles> <threads_per_block>
//
// =============================================================================
#include <hip/hip_runtime.h>
#include <iostream>
#include <random>
#include <hip/hip_runtime.h>
#include <math.h>
#include <numeric>
#include <algorithm>
#include <iterator>
#include <chrono>

#include "dof18_halfImplicit_gpu.cuh"

// A simple functor for demonstration
struct MyFunctor {
    // It is imperitve that these input outputs need to be maintained
    __device__ void operator()(double t, DriverInput* controls) {
        if (t < 1) {
            controls->m_steering = 0.0;
            controls->m_throttle = 0.0;
            controls->m_braking = 0.0;
        } else {
            controls->m_steering = 0.0;
            controls->m_throttle = 0.5;
            controls->m_braking = 0.0;
        }
    }
};

// Use ./executable_name <total_number_of_vehicles> <threads_per_block>

using namespace d18GPU;
int main(int argc, char** argv) {
    // Get total number of vehicles from command line
    unsigned int num_vehicles = std::stoul(argv[1]);
    // Set the threads per block from command line
    unsigned int threads_per_block = std::stoul(argv[2]);
    std::string file_name = "acc3";
    // Driver inputs -> All vehicles have the same driver inputs
    std::string driver_file = "../../18dof-gpu/data/input/" + file_name + ".txt";

    // Vehicle specification -> We assume that all vehicles have the same parameters
    std::string vehParamsJSON = (char*)"../../18dof-gpu/data/json/HMMWV/vehicle.json";
    std::string tireParamsJSON = (char*)"../../18dof-gpu/data/json/HMMWV/tmeasy.json";

    // Construct the solver
    d18SolverHalfImplicitGPU solver(num_vehicles);
    // The number of vehicles here sets these parameters and inputs for all these vehicles
    // If there is a need to set different parameters for different vehicles, then the solver
    // needs to be constructed for each vehicle separately (using the same sovler object)
    // No driver file
    solver.Construct(vehParamsJSON, tireParamsJSON, num_vehicles);

    // Set the threads per block
    solver.SetThreadsPerBlock(threads_per_block);

    // Set the time step of the solver
    solver.SetTimeStep(1e-3);

    // Decide on the "step" timestep and set it here
    double control_time_step = 1e-1;
    solver.SetKernelSimTime(control_time_step);

    // Now we initialize the states -> These are all set to 0 (struct initializer)
    VehicleState veh_st;
    TMeasyState tirelf_st;
    TMeasyState tirerf_st;
    TMeasyState tirelr_st;
    TMeasyState tirerr_st;
    // Again we initialize the same states for all vehicles
    solver.Initialize(veh_st, tirelf_st, tirerf_st, tirelr_st, tirerr_st, num_vehicles);

    // NOTE: SolveStep does not support storing output so both of these need to stay commented
    // solver.SetOutput("../../18dof-gpu/data/output/" + file_name + "_hmmwv18", 100, true);
    // Enable output for 50 of the vehicles
    // solver.SetOutput("../../18dof-gpu/data/output/" + file_name + "_hmmwv18step", 100, false, 50);

    // Initialize the controls functor
    MyFunctor myControlsFunctor;
    double endTime = 10.0;
    double timeStep = solver.GetStep();
    double t = 0;
    double new_time = 0;
    // Now solve in loop
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    while (t < (endTime - timeStep / 10.)) {
        new_time = solver.SolveStep(t, myControlsFunctor);  // Solve for the current time
        t = new_time;
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "Solve time (ms): " << milliseconds << "\n";
    // Extract terminal state of choosen vehicles and print the position
    SimState sim_state_1 = solver.GetSimState(0);

    std::cout << "X Position of vehicle 1: " << sim_state_1._veh_state._x << std::endl;
}
