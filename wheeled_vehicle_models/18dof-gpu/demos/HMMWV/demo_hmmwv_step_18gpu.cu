// =============================================================================
// Authors: Huzaifa Unjhawala
// =============================================================================
//
// This demo describes simulating user provided number of HMMWVs (specified with JSON files) on a step-by-step
// basis. The "step" at which a new control input is provided can be set with SetKernelSimTime as shown here.
// Since the Half Implicit solver is the only one supported for the GPU models, that is what is used here.
// When the solver is used in a step-by-step manner, the output is not stored in a file (unlike the CPU models).
// However, access to the vehicle states every control time step is provided through the GetSimState function.
// Use ./executable_name <total_number_of_vehicles> <threads_per_block>
//
// =============================================================================
#include <hip/hip_runtime.h>
#include <iostream>
#include <filesystem>
#include <hip/hip_runtime.h>
#include <math.h>
#include <numeric>
#include <algorithm>
#include <iterator>
#include <chrono>

#include "dof18_halfImplicit_gpu.cuh"

namespace fs = std::filesystem;
using namespace d18GPU;

int main(int argc, char** argv) {
    if (argc != 3) {
        std::cerr << "Usage: " << argv[0] << " <total_number_of_vehicles> <threads_per_block>" << std::endl;
        return 1;
    }

    unsigned int num_vehicles = std::stoul(argv[1]);
    unsigned int threads_per_block = std::stoul(argv[2]);
    std::string file_name = "acc3";
    std::string inputPath = "../../18dof-gpu/data/input/" + file_name + ".txt";

    // Ensure the input file exists
    if (!fs::exists(inputPath)) {
        std::cerr << "Error: Input file does not exist: " << inputPath << std::endl;
        return 1;
    }

    std::string vehParamsJSON = "../../18dof-gpu/data/json/HMMWV/vehicle.json";
    std::string tireParamsJSON = "../../18dof-gpu/data/json/HMMWV/tmeasy.json";

    // Construct the solver
    d18SolverHalfImplicitGPU solver(num_vehicles);
    solver.Construct(vehParamsJSON, tireParamsJSON, num_vehicles);

    solver.SetThreadsPerBlock(threads_per_block);
    solver.SetTimeStep(1e-3);
    solver.SetKernelSimTime(0.1);

    VehicleState veh_st;
    TMeasyState tirelf_st;
    TMeasyState tirerf_st;
    TMeasyState tirelr_st;
    TMeasyState tirerr_st;
    solver.Initialize(veh_st, tirelf_st, tirerf_st, tirelr_st, tirerr_st, num_vehicles);

    double endTime = 10.0;
    double timeStep = solver.GetStep();
    double t = 0;
    double new_time = 0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    double throttle;
    double steering;
    double braking;
    while (t < (endTime - timeStep / 10.)) {
        if (t > 1) {
            throttle = 0.5;
            steering = 0.0;
            braking = 0.0;
        } else {
            throttle = 0.0;
            steering = 0.0;
            braking = 0.0;
        }

        new_time = solver.SolveStep(t, steering, throttle, braking);  // Solve for the current time
        t = new_time;
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "Solve time (ms): " << milliseconds << "\n";

    SimState sim_state_1 = solver.GetSimState(0);
    std::cout << "X Position of vehicle 1: " << sim_state_1._veh_state._x << std::endl;
}
