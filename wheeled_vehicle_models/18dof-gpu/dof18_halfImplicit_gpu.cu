#include "hip/hip_runtime.h"
#include <algorithm>
#include <cassert>
#include <random>
#include <cmath>
#include <iostream>
#include <stdint.h>
#include <vector>

#include "dof18_gpu.cuh"
#include "dof18_halfImplicit_gpu.cuh"

using namespace d18GPU;
// ======================================================================================================================
__host__ d18SolverHalfImplicitGPU::d18SolverHalfImplicitGPU(unsigned int total_num_vehicles)
    : m_step(0.001),
      m_output(false),
      m_vehicle_count_tracker_params(0),
      m_vehicle_count_tracker_states(0),
      m_kernel_sim_time(2.),
      m_host_dump_time(10.),
      m_threads_per_block(32),
      m_tend(0.) {
    m_total_num_vehicles = total_num_vehicles;

    // Allocate memory for the simData and simStates
    CHECK_CUDA_ERROR(hipMallocManaged((void**)&m_sim_data, sizeof(d18GPU::SimData) * m_total_num_vehicles));
    CHECK_CUDA_ERROR(hipMallocManaged((void**)&m_sim_data_nr, sizeof(d18GPU::SimDataNr) * m_total_num_vehicles));
    CHECK_CUDA_ERROR(hipMallocManaged((void**)&m_sim_states, sizeof(d18GPU::SimState) * m_total_num_vehicles));
    CHECK_CUDA_ERROR(hipMallocManaged((void**)&m_sim_states_nr, sizeof(d18GPU::SimStateNr) * m_total_num_vehicles));

    // Set device and host arrays to nullptrs in case SetOutput is not called by the user
    m_device_response = nullptr;
    m_host_response = nullptr;

    int deviceId = 0;         // Assume we are using GPU 0
    hipSetDevice(deviceId);  // Set the device
}
__host__ d18SolverHalfImplicitGPU::~d18SolverHalfImplicitGPU() {
    // Only need to delete the memory of the simData and simStates of the respective tire as the rest of the memory is
    // freed as soon as we have information of what tire the user is using
    if (m_tire_type == TireType::TMeasy) {
        hipFree(m_sim_data);
        hipFree(m_sim_states);
    } else {
        hipFree(m_sim_data_nr);
        hipFree(m_sim_states_nr);
    }
    hipFree(m_device_response);
    delete[] m_host_response;
}
// ======================================================================================================================
// Construct the solver using path to vehicle parameters, tire parameters, number of vehicles and driver
__host__ void d18SolverHalfImplicitGPU::Construct(const std::string& vehicle_params_file,
                                                  const std::string& tire_params_file,
                                                  unsigned int num_vehicles,
                                                  const std::string& driver_inputs_file) {
    // Check if num_vehicles added is less than the total number of vehicles
    assert((num_vehicles + m_vehicle_count_tracker_params <= m_total_num_vehicles) &&
           "Number of vehicles added makes the vehicle count greater than the total number of vehicles");
    // If there is no tire type specified, then use TMeasy
    m_tire_type = TireType::TMeasy;
    // Because of this, we free the memory of the TMeasyNR tire
    hipFree(m_sim_data_nr);
    hipFree(m_sim_states_nr);
    // Set these to nullptr so that we don't try to free them again in the destructor
    m_sim_data_nr = nullptr;
    m_sim_states_nr = nullptr;

    // Since hipMallocManaged does not call the constructor for non-POD types, we create cpu structs and fill them up
    // and then copy them over to the simData structs
    d18GPU::VehicleParam veh_param;
    d18GPU::TMeasyParam tire_param;

    setVehParamsJSON(veh_param, vehicle_params_file.c_str());
    setTireParamsJSON(tire_param, tire_params_file.c_str());
    // Initialize tire parameters that depend on other parameters
    tireInit(&tire_param);

    DriverData driver_data;
    LoadDriverData(driver_data, driver_inputs_file);
    unsigned int driver_data_len = driver_data.size();
    size_t old_vehicle_count = m_vehicle_count_tracker_params;
    m_vehicle_count_tracker_params += num_vehicles;
    for (size_t i = old_vehicle_count; i < m_vehicle_count_tracker_params; i++) {
        m_sim_data[i]._driver_data_len = driver_data_len;
        // Allocate memory for the driver data
        CHECK_CUDA_ERROR(hipMallocManaged((void**)&m_sim_data[i]._driver_data,
                                           sizeof(DriverInput) * m_sim_data[i]._driver_data_len));
        // Copy the driver data from cpu to managed memory
        std::copy(driver_data.begin(), driver_data.end(), m_sim_data[i]._driver_data);
        // Fill up simulation data from the cpu structs
        m_sim_data[i]._veh_param = veh_param;
        m_sim_data[i]._tireTM_param = tire_param;
    }
    CHECK_CUDA_ERROR(hipMemPrefetchAsync(m_sim_data, sizeof(m_sim_data[0]) * m_vehicle_count_tracker_params,
                                          0));  // move the simData onto the GPU
}

__host__ void d18SolverHalfImplicitGPU::Construct(const std::string& vehicle_params_file,
                                                  const std::string& tire_params_file,
                                                  unsigned int num_vehicles,
                                                  const std::string& driver_inputs_file,
                                                  TireType type) {
    // Check if num_vehicles added is less than the total number of vehicles
    assert((num_vehicles + m_vehicle_count_tracker_params <= m_total_num_vehicles) &&
           "Number of vehicles added makes the vehicle count greater than the total number of vehicles");
    // If there is no tire type specified, then use TMeasy
    m_tire_type = type;

    if (m_tire_type == TireType::TMeasy) {
        // Because of this, we free the memory of the TMeasyNR tire
        hipFree(m_sim_data_nr);
        hipFree(m_sim_states_nr);
        // Set these to nullptr so that we don't try to free them again in the destructor
        m_sim_data_nr = nullptr;
        m_sim_states_nr = nullptr;

        // Since hipMallocManaged does not call the constructor for non-POD types, we create cpu structs and fill them
        // up and then copy them over to the simData structs
        d18GPU::VehicleParam veh_param;
        d18GPU::TMeasyParam tire_param;

        setVehParamsJSON(veh_param, vehicle_params_file.c_str());
        setTireParamsJSON(tire_param, tire_params_file.c_str());
        // Initialize tire parameters that depend on other parameters
        tireInit(&tire_param);

        DriverData driver_data;
        LoadDriverData(driver_data, driver_inputs_file);
        unsigned int driver_data_len = driver_data.size();
        size_t old_vehicle_count = m_vehicle_count_tracker_params;
        m_vehicle_count_tracker_params += num_vehicles;
        for (size_t i = old_vehicle_count; i < m_vehicle_count_tracker_params; i++) {
            m_sim_data[i]._driver_data_len = driver_data_len;
            // Allocate memory for the driver data
            CHECK_CUDA_ERROR(hipMallocManaged((void**)&m_sim_data[i]._driver_data,
                                               sizeof(DriverInput) * m_sim_data[i]._driver_data_len));
            // Copy the driver data from cpu to managed memory
            std::copy(driver_data.begin(), driver_data.end(), m_sim_data[i]._driver_data);
            // Fill up simulation data from the cpu structs
            m_sim_data[i]._veh_param = veh_param;
            m_sim_data[i]._tireTM_param = tire_param;
        }
        CHECK_CUDA_ERROR(hipMemPrefetchAsync(m_sim_data, sizeof(m_sim_data[0]) * m_vehicle_count_tracker_params,
                                              0));  // move the simData onto the GPU
    } else {
        // Because of this, we free the memory of the TMeasyNR tire
        hipFree(m_sim_data);
        hipFree(m_sim_states);
        // Set these to nullptr so that we don't try to free them again in the destructor
        m_sim_data = nullptr;
        m_sim_states = nullptr;

        // Since hipMallocManaged does not call the constructor for non-POD types, we create cpu structs and fill them
        // up and then copy them over to the simData structs
        d18GPU::VehicleParam veh_param;
        d18GPU::TMeasyNrParam tire_param;

        setVehParamsJSON(veh_param, vehicle_params_file.c_str());
        setTireParamsJSON(tire_param, tire_params_file.c_str());
        // Initialize tire parameters that depend on other parameters
        tireInit(&tire_param);

        DriverData driver_data;
        LoadDriverData(driver_data, driver_inputs_file);
        unsigned int driver_data_len = driver_data.size();
        size_t old_vehicle_count = m_vehicle_count_tracker_params;
        m_vehicle_count_tracker_params += num_vehicles;
        for (size_t i = old_vehicle_count; i < m_vehicle_count_tracker_params; i++) {
            m_sim_data_nr[i]._driver_data_len = driver_data_len;
            // Allocate memory for the driver data
            CHECK_CUDA_ERROR(hipMallocManaged((void**)&m_sim_data_nr[i]._driver_data,
                                               sizeof(DriverInput) * m_sim_data_nr[i]._driver_data_len));
            // Copy the driver data from cpu to managed memory
            std::copy(driver_data.begin(), driver_data.end(), m_sim_data_nr[i]._driver_data);
            // Fill up simulation data from the cpu structs
            m_sim_data_nr[i]._veh_param = veh_param;
            m_sim_data_nr[i]._tireTMNr_param = tire_param;
        }
        CHECK_CUDA_ERROR(hipMemPrefetchAsync(m_sim_data_nr, sizeof(m_sim_data_nr[0]) * m_vehicle_count_tracker_params,
                                              0));  // move the simData onto the GPU
    }
}

// Overload for situations when a controller is used and we don't have a driver data file
__host__ void d18SolverHalfImplicitGPU::Construct(const std::string& vehicle_params_file,
                                                  const std::string& tire_params_file,
                                                  unsigned int num_vehicles) {
    // Check if num_vehicles added is less than the total number of vehicles
    assert((num_vehicles + m_vehicle_count_tracker_params <= m_total_num_vehicles) &&
           "Number of vehicles added makes the vehicle count greater than the total number of vehicles");
    // If there is no tire type specified, then use TMeasy
    m_tire_type = TireType::TMeasy;
    // Because of this, we free the memory of the TMeasyNR tire
    hipFree(m_sim_data_nr);
    hipFree(m_sim_states_nr);
    // Set these to nullptr so that we don't try to free them again in the destructor
    m_sim_data_nr = nullptr;
    m_sim_states_nr = nullptr;

    // Since hipMallocManaged does not call the constructor for non-POD types, we create cpu structs and fill them up
    // and then copy them over to the simData structs
    d18GPU::VehicleParam veh_param;
    d18GPU::TMeasyParam tire_param;

    setVehParamsJSON(veh_param, vehicle_params_file.c_str());
    setTireParamsJSON(tire_param, tire_params_file.c_str());
    // Initialize tire parameters that depend on other parameters
    tireInit(&tire_param);

    size_t old_vehicle_count = m_vehicle_count_tracker_params;
    m_vehicle_count_tracker_params += num_vehicles;
    for (size_t i = old_vehicle_count; i < m_vehicle_count_tracker_params; i++) {
        // Fill up simulation data from the cpu structs
        m_sim_data[i]._veh_param = veh_param;
        m_sim_data[i]._tireTM_param = tire_param;
    }

    CHECK_CUDA_ERROR(hipMemPrefetchAsync(m_sim_data, sizeof(m_sim_data[0]) * m_vehicle_count_tracker_params,
                                          0));  // move the simData onto the GPU
}

__host__ void d18SolverHalfImplicitGPU::Construct(const std::string& vehicle_params_file,
                                                  const std::string& tire_params_file,
                                                  unsigned int num_vehicles,
                                                  TireType type) {
    // Check if num_vehicles added is less than the total number of vehicles
    assert((num_vehicles + m_vehicle_count_tracker_params <= m_total_num_vehicles) &&
           "Number of vehicles added makes the vehicle count greater than the total number of vehicles");
    m_tire_type = type;
    // If there is no tire type specified, then use TMeasy
    if (m_tire_type == TireType::TMeasy) {
        // Because of this, we free the memory of the TMeasyNR tire
        hipFree(m_sim_data_nr);
        hipFree(m_sim_states_nr);
        // Set these to nullptr so that we don't try to free them again in the destructor
        m_sim_data_nr = nullptr;
        m_sim_states_nr = nullptr;

        // Since hipMallocManaged does not call the constructor for non-POD types, we create cpu structs and fill them
        // up and then copy them over to the simData structs
        d18GPU::VehicleParam veh_param;
        d18GPU::TMeasyParam tire_param;

        setVehParamsJSON(veh_param, vehicle_params_file.c_str());
        setTireParamsJSON(tire_param, tire_params_file.c_str());
        // Initialize tire parameters that depend on other parameters
        tireInit(&tire_param);

        size_t old_vehicle_count = m_vehicle_count_tracker_params;
        m_vehicle_count_tracker_params += num_vehicles;
        for (size_t i = old_vehicle_count; i < m_vehicle_count_tracker_params; i++) {
            // Fill up simulation data from the cpu structs
            m_sim_data[i]._veh_param = veh_param;
            m_sim_data[i]._tireTM_param = tire_param;
        }
        CHECK_CUDA_ERROR(hipMemPrefetchAsync(m_sim_data, sizeof(m_sim_data[0]) * m_vehicle_count_tracker_params,
                                              0));  // move the simData onto the GPU
    } else {
        // Because of this, we free the memory of the TMeasyNR tire
        hipFree(m_sim_data);
        hipFree(m_sim_states);
        // Set these to nullptr so that we don't try to free them again in the destructor
        m_sim_data = nullptr;
        m_sim_states = nullptr;

        // Since hipMallocManaged does not call the constructor for non-POD types, we create cpu structs and fill them
        // up and then copy them over to the simData structs
        d18GPU::VehicleParam veh_param;
        d18GPU::TMeasyNrParam tire_param;

        setVehParamsJSON(veh_param, vehicle_params_file.c_str());
        setTireParamsJSON(tire_param, tire_params_file.c_str());
        // Initialize tire parameters that depend on other parameters
        tireInit(&tire_param);

        size_t old_vehicle_count = m_vehicle_count_tracker_params;
        m_vehicle_count_tracker_params += num_vehicles;
        for (size_t i = old_vehicle_count; i < m_vehicle_count_tracker_params; i++) {
            // Fill up simulation data from the cpu structs
            m_sim_data_nr[i]._veh_param = veh_param;
            m_sim_data_nr[i]._tireTMNr_param = tire_param;
        }
        CHECK_CUDA_ERROR(hipMemPrefetchAsync(m_sim_data_nr, sizeof(m_sim_data_nr[0]) * m_vehicle_count_tracker_params,
                                              0));  // move the simData onto the GPU
    }
}
// ======================================================================================================================

__host__ void d18SolverHalfImplicitGPU::Initialize(d18GPU::VehicleState& vehicle_states,
                                                   d18GPU::TMeasyState& tire_states_LF,
                                                   d18GPU::TMeasyState& tire_states_RF,
                                                   d18GPU::TMeasyState& tire_states_LR,
                                                   d18GPU::TMeasyState& tire_states_RR,
                                                   unsigned int num_vehicles) {
    // Ensure that construct was called with TMeasy tire type
    assert((m_tire_type == TireType::TMeasy) &&
           "Construct function called with TMeasyNr tire type, but Initialize called with TMeasy tire type");
    assert((num_vehicles + m_vehicle_count_tracker_states <= m_total_num_vehicles) &&
           "Number of vehicles added makes the vehicle count greater than the total number of vehicles");
    size_t old_vehicle_count = m_vehicle_count_tracker_states;
    m_vehicle_count_tracker_states += num_vehicles;
    for (size_t i = old_vehicle_count; i < m_vehicle_count_tracker_states; i++) {
        // Fill up simulation data from the cpu structs
        m_sim_states[i]._veh_state = vehicle_states;
        m_sim_states[i]._tirelf_state = tire_states_LF;
        m_sim_states[i]._tirerf_state = tire_states_RF;
        m_sim_states[i]._tirelr_state = tire_states_LR;
        m_sim_states[i]._tirerr_state = tire_states_RR;
    }
    CHECK_CUDA_ERROR(hipMemPrefetchAsync(m_sim_states, sizeof(SimState) * m_vehicle_count_tracker_states,
                                          0));  // move the simState onto the GPU
}

// TMeasy without relaxation does not have tire states and so the jacobian size reduces by 8
__host__ void d18SolverHalfImplicitGPU::Initialize(d18GPU::VehicleState& vehicle_states,
                                                   d18GPU::TMeasyNrState& tire_states_LF,
                                                   d18GPU::TMeasyNrState& tire_states_RF,
                                                   d18GPU::TMeasyNrState& tire_states_LR,
                                                   d18GPU::TMeasyNrState& tire_states_RR,
                                                   unsigned int num_vehicles) {
    // Ensure that construct was called with TMeasyNr tire type
    assert((m_tire_type == TireType::TMeasyNr) &&
           "Construct function called with TMeasy tire type, but Initialize called with TMeasyNR tire type");
    assert((num_vehicles + m_vehicle_count_tracker_states <= m_total_num_vehicles) &&
           "Number of vehicles added makes the vehicle count greater than the total number of vehicles");
    size_t old_vehicle_count = m_vehicle_count_tracker_states;
    m_vehicle_count_tracker_states += num_vehicles;
    for (size_t i = old_vehicle_count; i < m_vehicle_count_tracker_states; i++) {
        // Fill up simulation data from the cpu structs
        m_sim_states_nr[i]._veh_state = vehicle_states;
        m_sim_states_nr[i]._tirelf_state = tire_states_LF;
        m_sim_states_nr[i]._tirerf_state = tire_states_RF;
        m_sim_states_nr[i]._tirelr_state = tire_states_LR;
        m_sim_states_nr[i]._tirerr_state = tire_states_RR;
    }
    CHECK_CUDA_ERROR(hipMemPrefetchAsync(m_sim_states_nr, sizeof(SimState) * m_vehicle_count_tracker_states,
                                          0));  // move the simState onto the GPU
}

// ======================================================================================================================
__host__ void d18SolverHalfImplicitGPU::SetOutput(const std::string& output_file,
                                                  double output_freq,
                                                  bool store_all,
                                                  unsigned int no_outs) {
    m_output = true;
    m_store_all = store_all;
    if (!m_store_all) {
        // Check if number of outputs asked is greater than the total number of vehicles, if this is the case, raise
        // awarning and set to m_total_num_vehicles
        if (no_outs > m_total_num_vehicles) {
            std::cout << "Number of outputs asked is greater than the total number of vehicles, setting number of "
                         "outputs to total number of vehicles"
                      << std::endl;
            no_outs = m_total_num_vehicles;
        }
        m_num_outs = no_outs;
        // If store_all is false, randomly assign which vehicles need to be dumped into csv
        float some_seed = 68;
        std::mt19937 generator(some_seed);

        // Generate a range of numbers and shuffle them
        std::vector<int> numbers(m_total_num_vehicles);
        std::iota(numbers.begin(), numbers.end(), 0);  // Fill with values from 0 to m_total_num_vehicles - 1
        std::shuffle(numbers.begin(), numbers.end(), generator);

        // Resize m_which_outs and assign the first 'no_outs' numbers from the shuffled range
        m_which_outs.resize(no_outs);
        std::copy(numbers.begin(), numbers.begin() + no_outs, m_which_outs.begin());
    } else {
        m_num_outs = m_total_num_vehicles;
    }
    // Allocate memory for the csv_writers
    m_csv_writers_ptr = std::make_unique<CSV_writer[]>(m_num_outs);
    m_output_file = output_file;
    m_dtout = 1.0 / output_freq;

    // Here we also initialize the device and host arrays that store the response across vehicles and states -> If
    // output is not required, then nothing is stored, however the user has access to the states at the last time step
    // through the simState and simData structs

    // Number of time steps to be collected on the device
    m_device_collection_timeSteps = ceil(m_kernel_sim_time / m_dtout);

    // Number of states to store -> For now we only allow storage of the major states which are common to both tire
    // models [time,x,y,u,v,phi,psi,wx,wz,lf_omega,rf_omega,lr_omega,rr_omega]
    m_collection_states = 13;

    // Thus device array size becomes
    m_device_size = sizeof(double) * m_total_num_vehicles * m_collection_states * (m_device_collection_timeSteps);

    CHECK_CUDA_ERROR(hipMalloc((void**)&m_device_response, m_device_size));

    // Now the host response
    m_host_collection_timeSteps = ceil(m_host_dump_time / m_dtout);

    // Thus the host size becomes -> Usually much larger than the device size
    m_host_response = new double[m_total_num_vehicles * m_collection_states * (m_host_collection_timeSteps)]();
}

// ======================================================================================================================

/// @brief Solve the system of equations by calling the integrate function
__host__ void d18SolverHalfImplicitGPU::Solve() {
    assert(m_tend != 0. && "Final time not set, please use SetEndTime function");
    // Calculate the number of blocks required
    m_num_blocks = (m_total_num_vehicles + m_threads_per_block - 1) / m_threads_per_block;
#ifdef DEBUG
    std::cout << "Number of blocks: " << m_num_blocks << std::endl;
    std::cout << "Number of threads per block: " << m_threads_per_block << std::endl;
    std::cout << "Total number of vehicles: " << m_total_num_vehicles << std::endl;
#endif
    // If m_output is false, then we still need to initialize device array -> We don't need the host array as its
    // purpose is just to store the final output
    if (!m_output) {
        // Number of time steps to be collected on the device
        m_device_collection_timeSteps = ceil(m_kernel_sim_time / m_dtout);

        // Number of states to store -> For now we only allow storage of the major states which are common to both tire
        // models [time,x,y,u,v,phi,psi,wx,wz,lf_omega,rf_omega,lr_omega,rr_omega]
        m_collection_states = 13;

        // Thus device array size becomes
        m_device_size = sizeof(double) * m_total_num_vehicles * m_collection_states * (m_device_collection_timeSteps);

        CHECK_CUDA_ERROR(hipMalloc((void**)&m_device_response, m_device_size));
    }

    double current_time = 0.;
    unsigned int kernel_launches_since_last_dump = 0;  // Track the number of kernel launches since the last dump of the
                                                       // host response
    double time_since_last_dump = 0.;                  // Track the time since the last dump of the host response
    // Write the initial conditions
    if (m_output) {
        Write(current_time);
    }

    while (current_time < m_tend) {
        // Calculate when this kernel is supposed to end

        double kernel_end_time = current_time + m_kernel_sim_time;

        // Launch the kernel
        if (m_tire_type == TireType::TMeasy) {
            Integrate<<<m_num_blocks, m_threads_per_block>>>(current_time, m_kernel_sim_time, m_step, m_output,
                                                             m_total_num_vehicles, m_collection_states, m_dtout,
                                                             m_device_response, m_sim_data, m_sim_states);
            hipError_t err = hipGetLastError();
            if (err != hipSuccess) {
                fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(err));
            }
        } else {
            Integrate<<<m_num_blocks, m_threads_per_block>>>(current_time, m_kernel_sim_time, m_step, m_output,
                                                             m_total_num_vehicles, m_collection_states, m_dtout,
                                                             m_device_response, m_sim_data_nr, m_sim_states_nr);
            hipError_t err = hipGetLastError();
            if (err != hipSuccess) {
                fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(err));
            }
        }

        // Get the new time the simulation has reached
        current_time = kernel_end_time;
        time_since_last_dump += m_kernel_sim_time;

        // If we have to save output, copy over the device into the response
        if (m_output) {
            // Amount of respoonse already filled
            unsigned int filled_response = m_total_num_vehicles * m_collection_states * m_device_collection_timeSteps *
                                           kernel_launches_since_last_dump;

            // Copy the device to the right part of the host response
            CHECK_CUDA_ERROR(hipMemcpy(m_host_response + filled_response, m_device_response, m_device_size,
                                        hipMemcpyDeviceToHost));

            kernel_launches_since_last_dump++;
            // Check if host is full and dump that into a csv writer
            if (abs(time_since_last_dump - m_host_dump_time) < 1e-6) {
                Write(current_time);
                time_since_last_dump = 0.;
                kernel_launches_since_last_dump = 0;
            }
        }
    }
    // If the simulation ended at a non multiple of m_host_dump_time, we dump the remaining data
    if (m_output && (kernel_launches_since_last_dump != 0)) {
        unsigned int time_steps_to_write = kernel_launches_since_last_dump * m_device_collection_timeSteps;
        Write(current_time, time_steps_to_write);
    }
    // End of simulation, write to the csv file
    if (m_output) {
        WriteToFile();
    }
}

//======================================================================================================================
__host__ double d18SolverHalfImplicitGPU::SolveStep(double t,
                                                    double steering,
                                                    double throttle,
                                                    double braking) {  // Calculate the number of blocks required
    // if m_output is true, then raise assertion
    if (m_output) {
        // Handle the error: log, return an error code, etc.
        std::cerr << "Cannot get csv file output if SolveStep is called, please access sim_states through GetSimSate"
                  << std::endl;
        exit(EXIT_FAILURE);
    }
    m_num_blocks = (m_total_num_vehicles + m_threads_per_block - 1) / m_threads_per_block;
    // If m_output is false and its the first time step then we still need to initialize device array -> We don't need
    // the host array as its purpose is just to store the final output
    if (t == 0.) {
        // Number of time steps to be collected on the device
        m_device_collection_timeSteps = ceil(m_kernel_sim_time / m_dtout);

        // Number of states to store -> For now we only allow storage of the major states which are common to both tire
        // models [time,x,y,u,v,phi,psi,wx,wz,lf_omega,rf_omega,lr_omega,rr_omega]
        m_collection_states = 13;

        // Thus device array size becomes
        m_device_size = sizeof(double) * m_total_num_vehicles * m_collection_states * (m_device_collection_timeSteps);

        CHECK_CUDA_ERROR(hipMalloc((void**)&m_device_response, m_device_size));
    }
    m_current_time = t;

    // Launch the kernel
    double kernel_end_time = m_current_time + m_kernel_sim_time;

    if (m_tire_type == TireType::TMeasy) {
        Integrate<<<m_num_blocks, m_threads_per_block>>>(m_current_time, steering, throttle, braking, m_kernel_sim_time,
                                                         m_step, m_output, m_total_num_vehicles, m_collection_states,
                                                         m_dtout, m_device_response, m_sim_data, m_sim_states);
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(err));
        }
    } else {
        Integrate<<<m_num_blocks, m_threads_per_block>>>(m_current_time, steering, throttle, braking, m_kernel_sim_time,
                                                         m_step, m_output, m_total_num_vehicles, m_collection_states,
                                                         m_dtout, m_device_response, m_sim_data_nr, m_sim_states_nr);
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(err));
        }
    }

    m_current_time = kernel_end_time;
    m_time_since_last_dump += m_kernel_sim_time;
    return m_current_time;
}
//======================================================================================================================
__host__ void d18SolverHalfImplicitGPU::Write(double t, unsigned int time_steps_to_write) {
    unsigned int loop_limit = 0;
    if (m_store_all) {
        loop_limit = m_total_num_vehicles;
    } else {
        loop_limit = m_num_outs;
    }

    // If time_steps_to_write is not specified, we write all the data
    if (time_steps_to_write == 0) {
        time_steps_to_write = m_host_collection_timeSteps;
    }

    if (t < m_step) {
        for (unsigned int sim_no = 0; sim_no < loop_limit; sim_no++) {
            CSV_writer& csv = m_csv_writers_ptr[sim_no];
            csv << "time";
            csv << "x";
            csv << "y";
            csv << "vx";
            csv << "vy";
            csv << "roll";
            csv << "yaw";
            csv << "roll_rate";
            csv << "yaw_rate";
            csv << "wlf";
            csv << "wrf";
            csv << "wlr";
            csv << "wrr";
            csv << std::endl;

            csv << 0;
            csv << 0;
            csv << 0;
            csv << 0;
            csv << 0;
            csv << 0;
            csv << 0;
            csv << 0;
            csv << 0;
            csv << 0;
            csv << 0;
            csv << 0;
            csv << 0;
            csv << std::endl;
        }
        return;
    } else {
        for (unsigned int sim_no = 0; sim_no < loop_limit; sim_no++) {
            unsigned int index_by = 0;
            // If we are no storing all, we will have to index by random numbers
            if (m_store_all) {
                index_by = sim_no;
            } else {
                index_by = m_which_outs[sim_no];
            }
            CSV_writer& csv = m_csv_writers_ptr[sim_no];
            unsigned int steps_written = 0;
            while (steps_written < time_steps_to_write) {
                unsigned int time_offset = steps_written * m_total_num_vehicles * m_collection_states;
                csv << m_host_response[time_offset + (m_total_num_vehicles * 0) + index_by];
                csv << m_host_response[time_offset + (m_total_num_vehicles * 1) + index_by];
                csv << m_host_response[time_offset + (m_total_num_vehicles * 2) + index_by];
                csv << m_host_response[time_offset + (m_total_num_vehicles * 3) + index_by];
                csv << m_host_response[time_offset + (m_total_num_vehicles * 4) + index_by];
                csv << m_host_response[time_offset + (m_total_num_vehicles * 5) + index_by];
                csv << m_host_response[time_offset + (m_total_num_vehicles * 6) + index_by];
                csv << m_host_response[time_offset + (m_total_num_vehicles * 7) + index_by];
                csv << m_host_response[time_offset + (m_total_num_vehicles * 8) + index_by];
                csv << m_host_response[time_offset + (m_total_num_vehicles * 9) + index_by];
                csv << m_host_response[time_offset + (m_total_num_vehicles * 10) + index_by];
                csv << m_host_response[time_offset + (m_total_num_vehicles * 11) + index_by];
                csv << m_host_response[time_offset + (m_total_num_vehicles * 12) + index_by];
                csv << std::endl;
                steps_written++;
            }
        }
    }
}

// ======================================================================================================================

__host__ void d18SolverHalfImplicitGPU::WriteToFile() {
    if (!m_output) {
        std::cout << "No output file specified. Call SetOutput() before calling WriteToFile()" << std::endl;
        return;
    }
    unsigned int loop_limit = 0;
    if (m_store_all) {
        loop_limit = m_total_num_vehicles;
    } else {
        loop_limit = m_num_outs;
    }
    for (unsigned int sim_no = 0; sim_no < loop_limit; sim_no++) {
        unsigned int index_by = 0;
        // If we are not storing all, we will have to index by random numbers
        if (m_store_all) {
            index_by = sim_no;
        } else {
            index_by = m_which_outs[sim_no];
        }
        if (m_tire_type == TireType::TMeasy) {
            CSV_writer& csv = m_csv_writers_ptr[sim_no];
            csv.write_to_file(m_output_file + "_" + std::to_string(index_by) + ".csv");
            csv.clearData();
        } else {
            CSV_writer& csv = m_csv_writers_ptr[sim_no];
            csv.write_to_file(m_output_file + "_" + std::to_string(index_by) + ".csv");
            csv.clearData();
        }
    }
}

// ======================================================================================================================

__host__ SimState d18SolverHalfImplicitGPU::GetSimState(unsigned int vehicle_index) {
    assert((vehicle_index < m_total_num_vehicles) && "Vehicle index out of bounds");

    // Allocate space for a single SimState on the host
    SimState host_state;

    if (m_tire_type == TireType::TMeasy) {
        // Copy the specific SimState from the GPU to the host
        hipMemcpy(&host_state, &m_sim_states[vehicle_index], sizeof(SimState), hipMemcpyDeviceToHost);
    } else {
        // Similarly for m_sim_states_nr
        hipMemcpy(&host_state, &m_sim_states_nr[vehicle_index], sizeof(SimState), hipMemcpyDeviceToHost);
    }

    return host_state;
}

// ======================================================================================================================
__device__ void rhsFun(double t, unsigned int total_num_vehicles, SimData* sim_data, SimState* sim_states) {
    // Get the vehicle index
    unsigned int vehicle_index = blockIdx.x * blockDim.x + threadIdx.x;

    if (vehicle_index < total_num_vehicles) {
        // All vehicles have one or the other tire type and thus no thread divergence
        VehicleParam& veh_param = sim_data[vehicle_index]._veh_param;
        VehicleState& veh_state = sim_states[vehicle_index]._veh_state;
        TMeasyParam& tireTM_param = sim_data[vehicle_index]._tireTM_param;
        TMeasyState& tireTMlf_state = sim_states[vehicle_index]._tirelf_state;
        TMeasyState& tireTMrf_state = sim_states[vehicle_index]._tirerf_state;
        TMeasyState& tireTMlr_state = sim_states[vehicle_index]._tirelr_state;
        TMeasyState& tireTMrr_state = sim_states[vehicle_index]._tirerr_state;

        DriverInput* driver_data = sim_data[vehicle_index]._driver_data;
        unsigned int len = sim_data[vehicle_index]._driver_data_len;
        // Get controls at the current timeStep
        auto controls = GetDriverInput(t, driver_data, len);

        double loads[4] = {0., 0., 0., 0.};
        // Compute the tire loads
        computeTireLoads(&loads[0], &veh_state, &veh_param, &tireTM_param);
        // Transform from vehicle frame to the tire frame
        vehToTireTransform(&tireTMlf_state, &tireTMrf_state, &tireTMlr_state, &tireTMrr_state, &veh_state, &loads[0],
                           &veh_param, controls.m_steering);

        // Tire velocities using TMEasy tire
        computeTireRHS(&tireTMlf_state, &tireTM_param, &veh_param, controls.m_steering);
        computeTireRHS(&tireTMrf_state, &tireTM_param, &veh_param, controls.m_steering);
        computeTireRHS(&tireTMlr_state, &tireTM_param, &veh_param, 0);  // No rear steering
        computeTireRHS(&tireTMrr_state, &tireTM_param, &veh_param, 0);  // No rear steering

        // Powertrain dynamics
        computePowertrainRHS(&veh_state, &tireTMlf_state, &tireTMrf_state, &tireTMlr_state, &tireTMrr_state, &veh_param,
                             &tireTM_param, &controls);
        // Vehicle dynamics
        tireToVehTransform(&tireTMlf_state, &tireTMrf_state, &tireTMlr_state, &tireTMrr_state, &veh_state, &veh_param,
                           controls.m_steering);

        double fx[4] = {tireTMlf_state._fx, tireTMrf_state._fx, tireTMlr_state._fx, tireTMrr_state._fx};
        double fy[4] = {tireTMlf_state._fy, tireTMrf_state._fy, tireTMlr_state._fy, tireTMrr_state._fy};

        computeVehRHS(&veh_state, &veh_param, &fx[0], &fy[0]);
    }
}
// ======================================================================================================================
__device__ void rhsFun(double t,
                       unsigned int total_num_vehicles,
                       d18GPU::SimData* sim_data,
                       d18GPU::SimState* sim_states,
                       double steering,
                       double throttle,
                       double braking) {
    // Get the vehicle index
    unsigned int vehicle_index = blockIdx.x * blockDim.x + threadIdx.x;

    if (vehicle_index < total_num_vehicles) {
        // All vehicles have one or the other tire type and thus no thread divergence
        VehicleParam& veh_param = sim_data[vehicle_index]._veh_param;
        VehicleState& veh_state = sim_states[vehicle_index]._veh_state;
        TMeasyParam& tireTM_param = sim_data[vehicle_index]._tireTM_param;
        TMeasyState& tireTMlf_state = sim_states[vehicle_index]._tirelf_state;
        TMeasyState& tireTMrf_state = sim_states[vehicle_index]._tirerf_state;
        TMeasyState& tireTMlr_state = sim_states[vehicle_index]._tirelr_state;
        TMeasyState& tireTMrr_state = sim_states[vehicle_index]._tirerr_state;

        // Get controls at the current timeStep
        DriverInput controls;
        controls.m_steering = steering;
        controls.m_throttle = throttle;
        controls.m_braking = braking;

        double loads[4] = {0., 0., 0., 0.};
        // Compute the tire loads
        computeTireLoads(&loads[0], &veh_state, &veh_param, &tireTM_param);
        // Transform from vehicle frame to the tire frame
        vehToTireTransform(&tireTMlf_state, &tireTMrf_state, &tireTMlr_state, &tireTMrr_state, &veh_state, &loads[0],
                           &veh_param, controls.m_steering);

        // Tire velocities using TMEasy tire
        computeTireRHS(&tireTMlf_state, &tireTM_param, &veh_param, controls.m_steering);
        computeTireRHS(&tireTMrf_state, &tireTM_param, &veh_param, controls.m_steering);
        computeTireRHS(&tireTMlr_state, &tireTM_param, &veh_param, 0);  // No rear steering
        computeTireRHS(&tireTMrr_state, &tireTM_param, &veh_param, 0);  // No rear steering

        // Powertrain dynamics
        computePowertrainRHS(&veh_state, &tireTMlf_state, &tireTMrf_state, &tireTMlr_state, &tireTMrr_state, &veh_param,
                             &tireTM_param, &controls);
        // Vehicle dynamics
        tireToVehTransform(&tireTMlf_state, &tireTMrf_state, &tireTMlr_state, &tireTMrr_state, &veh_state, &veh_param,
                           controls.m_steering);

        double fx[4] = {tireTMlf_state._fx, tireTMrf_state._fx, tireTMlr_state._fx, tireTMrr_state._fx};
        double fy[4] = {tireTMlf_state._fy, tireTMrf_state._fy, tireTMlr_state._fy, tireTMrr_state._fy};

        computeVehRHS(&veh_state, &veh_param, &fx[0], &fy[0]);
    }
}
// ======================================================================================================================
__device__ void rhsFun(double t, unsigned int total_num_vehicles, SimDataNr* sim_data_nr, SimStateNr* sim_states_nr) {
    // Get the vehicle index
    unsigned int vehicle_index = blockIdx.x * blockDim.x + threadIdx.x;

    if (vehicle_index < total_num_vehicles) {
        VehicleParam& veh_param = sim_data_nr[vehicle_index]._veh_param;
        VehicleState& veh_state = sim_states_nr[vehicle_index]._veh_state;
        TMeasyNrParam& tireTMNr_param = sim_data_nr[vehicle_index]._tireTMNr_param;
        TMeasyNrState& tireTMNrlf_state = sim_states_nr[vehicle_index]._tirelf_state;
        TMeasyNrState& tireTMNrrf_state = sim_states_nr[vehicle_index]._tirerf_state;
        TMeasyNrState& tireTMNrlr_state = sim_states_nr[vehicle_index]._tirelr_state;
        TMeasyNrState& tireTMNrrr_state = sim_states_nr[vehicle_index]._tirerr_state;
        DriverInput* driver_data = sim_data_nr[vehicle_index]._driver_data;
        unsigned int len = sim_data_nr[vehicle_index]._driver_data_len;
        // Get controls at the current timeStep
        auto controls = GetDriverInput(t, driver_data, len);

        double loads[4] = {0., 0., 0., 0.};

        // Compute the tire loads
        computeTireLoads(&loads[0], &veh_state, &veh_param, &tireTMNr_param);
        // Transform from vehicle frame to the tire frame
        vehToTireTransform(&tireTMNrlf_state, &tireTMNrrf_state, &tireTMNrlr_state, &tireTMNrrr_state, &veh_state,
                           &loads[0], &veh_param, controls.m_steering);
        // Tire velocities using TMEasyNr tire
        computeTireRHS(&tireTMNrlf_state, &tireTMNr_param, &veh_param, controls.m_steering);
        computeTireRHS(&tireTMNrrf_state, &tireTMNr_param, &veh_param, controls.m_steering);
        computeTireRHS(&tireTMNrlr_state, &tireTMNr_param, &veh_param, 0);  // No rear steering
        computeTireRHS(&tireTMNrrr_state, &tireTMNr_param, &veh_param, 0);  // No rear steering

        // Powertrain dynamics
        computePowertrainRHS(&veh_state, &tireTMNrlf_state, &tireTMNrrf_state, &tireTMNrlr_state, &tireTMNrrr_state,
                             &veh_param, &tireTMNr_param, &controls);

        // Vehicle dynamics
        tireToVehTransform(&tireTMNrlf_state, &tireTMNrrf_state, &tireTMNrlr_state, &tireTMNrrr_state, &veh_state,
                           &veh_param, controls.m_steering);

        double fx[4] = {tireTMNrlf_state._fx, tireTMNrrf_state._fx, tireTMNrlr_state._fx, tireTMNrrr_state._fx};
        double fy[4] = {tireTMNrlf_state._fy, tireTMNrrf_state._fy, tireTMNrlr_state._fy, tireTMNrrr_state._fy};

        computeVehRHS(&veh_state, &veh_param, &fx[0], &fy[0]);
    }
}
// ======================================================================================================================
__device__ void rhsFun(double t,
                       unsigned int total_num_vehicles,
                       d18GPU::SimDataNr* sim_data_nr,
                       d18GPU::SimStateNr* sim_states_nr,
                       double steering,
                       double throttle,
                       double braking) {
    // Get the vehicle index
    unsigned int vehicle_index = blockIdx.x * blockDim.x + threadIdx.x;

    if (vehicle_index < total_num_vehicles) {
        VehicleParam& veh_param = sim_data_nr[vehicle_index]._veh_param;
        VehicleState& veh_state = sim_states_nr[vehicle_index]._veh_state;
        TMeasyNrParam& tireTMNr_param = sim_data_nr[vehicle_index]._tireTMNr_param;
        TMeasyNrState& tireTMNrlf_state = sim_states_nr[vehicle_index]._tirelf_state;
        TMeasyNrState& tireTMNrrf_state = sim_states_nr[vehicle_index]._tirerf_state;
        TMeasyNrState& tireTMNrlr_state = sim_states_nr[vehicle_index]._tirelr_state;
        TMeasyNrState& tireTMNrrr_state = sim_states_nr[vehicle_index]._tirerr_state;

        DriverInput controls;
        controls.m_steering = steering;
        controls.m_throttle = throttle;
        controls.m_braking = braking;

        double loads[4] = {0., 0., 0., 0.};

        // Compute the tire loads
        computeTireLoads(&loads[0], &veh_state, &veh_param, &tireTMNr_param);
        // Transform from vehicle frame to the tire frame
        vehToTireTransform(&tireTMNrlf_state, &tireTMNrrf_state, &tireTMNrlr_state, &tireTMNrrr_state, &veh_state,
                           &loads[0], &veh_param, controls.m_steering);
        // Tire velocities using TMEasyNr tire
        computeTireRHS(&tireTMNrlf_state, &tireTMNr_param, &veh_param, controls.m_steering);
        computeTireRHS(&tireTMNrrf_state, &tireTMNr_param, &veh_param, controls.m_steering);
        computeTireRHS(&tireTMNrlr_state, &tireTMNr_param, &veh_param, 0);  // No rear steering
        computeTireRHS(&tireTMNrrr_state, &tireTMNr_param, &veh_param, 0);  // No rear steering

        // Powertrain dynamics
        computePowertrainRHS(&veh_state, &tireTMNrlf_state, &tireTMNrrf_state, &tireTMNrlr_state, &tireTMNrrr_state,
                             &veh_param, &tireTMNr_param, &controls);

        // Vehicle dynamics
        tireToVehTransform(&tireTMNrlf_state, &tireTMNrrf_state, &tireTMNrlr_state, &tireTMNrrr_state, &veh_state,
                           &veh_param, controls.m_steering);

        double fx[4] = {tireTMNrlf_state._fx, tireTMNrrf_state._fx, tireTMNrlr_state._fx, tireTMNrrr_state._fx};
        double fy[4] = {tireTMNrlf_state._fy, tireTMNrrf_state._fy, tireTMNrlr_state._fy, tireTMNrrr_state._fy};

        computeVehRHS(&veh_state, &veh_param, &fx[0], &fy[0]);
    }
}
//======================================================================================================================

__global__ void Integrate(double current_time,
                          double kernel_sim_time,
                          double step,
                          bool output,
                          unsigned int total_num_vehicles,
                          unsigned int collection_states,
                          double dtout,
                          double* device_response,
                          SimData* sim_data,
                          SimState* sim_states) {
    double t = current_time;           // Set the current time
    double kernel_time = 0;            // Time since kernel was launched
    unsigned int timeStep_stored = 0;  // Number of time steps already stored in the device response
    double end_time = (t + kernel_sim_time) - step / 10.;
    unsigned int vehicle_id = blockIdx.x * blockDim.x + threadIdx.x;  // Get the vehicle id
    if (vehicle_id < total_num_vehicles) {
        while (t < end_time) {
            // Call the RHS to get accelerations for all the vehicles
            rhsFun(t, total_num_vehicles, sim_data, sim_states);

            // Integrate according to half implicit method for second order states
            // Integrate according to explicit method for first order states

            // Extract the states of the vehicle and the tires
            VehicleState& v_states = sim_states[vehicle_id]._veh_state;
            VehicleParam& veh_param = sim_data[vehicle_id]._veh_param;
            TMeasyState& tirelf_st = sim_states[vehicle_id]._tirelf_state;
            TMeasyState& tirerf_st = sim_states[vehicle_id]._tirerf_state;
            TMeasyState& tirelr_st = sim_states[vehicle_id]._tirelr_state;
            TMeasyState& tirerr_st = sim_states[vehicle_id]._tirerr_state;

            // First the tire states
            // LF
            tirelf_st._xe += tirelf_st._xedot * step;
            tirelf_st._ye += tirelf_st._yedot * step;
            tirelf_st._omega += tirelf_st._dOmega * step;
            // RF
            tirerf_st._xe += tirerf_st._xedot * step;
            tirerf_st._ye += tirerf_st._yedot * step;
            tirerf_st._omega += tirerf_st._dOmega * step;
            // LR
            tirelr_st._xe += tirelr_st._xedot * step;
            tirelr_st._ye += tirelr_st._yedot * step;
            tirelr_st._omega += tirelr_st._dOmega * step;
            // RR
            tirerr_st._xe += tirerr_st._xedot * step;
            tirerr_st._ye += tirerr_st._yedot * step;
            tirerr_st._omega += tirerr_st._dOmega * step;

            // Now the vehicle states
            if (veh_param._tcbool) {
                v_states._crankOmega += v_states._dOmega_crank * step;
            }

            // Integrate velocity level first
            v_states._u += v_states._udot * step;
            v_states._v += v_states._vdot * step;
            v_states._wx += v_states._wxdot * step;
            v_states._wz += v_states._wzdot * step;

            // Integrate position level next
            v_states._x += (v_states._u * cos(v_states._psi) - v_states._v * sin(v_states._psi)) * step;
            v_states._y += (v_states._u * sin(v_states._psi) + v_states._v * cos(v_states._psi)) * step;
            v_states._psi += v_states._wz * step;
            v_states._phi += v_states._wx * step;

            // Update time
            t += step;
            kernel_time += step;

            // Write to response if required -> regardless of no_outs or store_all we write all the vehicles to the
            // response
            if (output) {
                // The +1 here is because state at time 0 is not stored in device response
                if (abs(kernel_time - (timeStep_stored + 1) * dtout) < 1e-7) {
                    unsigned int time_offset = timeStep_stored * total_num_vehicles * collection_states;

                    device_response[time_offset + (total_num_vehicles * 0) + vehicle_id] = t;
                    device_response[time_offset + (total_num_vehicles * 1) + vehicle_id] = v_states._x;
                    device_response[time_offset + (total_num_vehicles * 2) + vehicle_id] = v_states._y;
                    device_response[time_offset + (total_num_vehicles * 3) + vehicle_id] = v_states._u;
                    device_response[time_offset + (total_num_vehicles * 4) + vehicle_id] = v_states._v;
                    device_response[time_offset + (total_num_vehicles * 5) + vehicle_id] = v_states._phi;
                    device_response[time_offset + (total_num_vehicles * 6) + vehicle_id] = v_states._psi;
                    device_response[time_offset + (total_num_vehicles * 7) + vehicle_id] = v_states._wx;
                    device_response[time_offset + (total_num_vehicles * 8) + vehicle_id] = v_states._wz;
                    device_response[time_offset + (total_num_vehicles * 9) + vehicle_id] = tirelf_st._omega;
                    device_response[time_offset + (total_num_vehicles * 10) + vehicle_id] = tirerf_st._omega;
                    device_response[time_offset + (total_num_vehicles * 11) + vehicle_id] = tirelr_st._omega;
                    device_response[time_offset + (total_num_vehicles * 12) + vehicle_id] = tirerr_st._omega;
                    timeStep_stored++;
                }
            }
        }
    }
}
// ======================================================================================================================
__global__ void Integrate(double current_time,
                          double steering,
                          double throttle,
                          double braking,
                          double kernel_sim_time,
                          double step,
                          bool output,
                          unsigned int total_num_vehicles,
                          unsigned int collection_states,
                          double dtout,
                          double* device_response,
                          d18GPU::SimData* sim_data,
                          d18GPU::SimState* sim_states) {
    double t = current_time;           // Set the current time
    double kernel_time = 0;            // Time since kernel was launched
    unsigned int timeStep_stored = 0;  // Number of time steps already stored in the device response
    double end_time = (t + kernel_sim_time) - step / 10.;
    unsigned int vehicle_id = blockIdx.x * blockDim.x + threadIdx.x;  // Get the vehicle id
    if (vehicle_id < total_num_vehicles) {
        while (t < end_time) {
            // Call the RHS to get accelerations for all the vehicles
            rhsFun(t, total_num_vehicles, sim_data, sim_states, steering, throttle, braking);

            // Integrate according to half implicit method for second order states
            // Integrate according to explicit method for first order states

            // Extract the states of the vehicle and the tires
            VehicleState& v_states = sim_states[vehicle_id]._veh_state;
            VehicleParam& veh_param = sim_data[vehicle_id]._veh_param;
            TMeasyState& tirelf_st = sim_states[vehicle_id]._tirelf_state;
            TMeasyState& tirerf_st = sim_states[vehicle_id]._tirerf_state;
            TMeasyState& tirelr_st = sim_states[vehicle_id]._tirelr_state;
            TMeasyState& tirerr_st = sim_states[vehicle_id]._tirerr_state;

            // First the tire states
            // LF
            tirelf_st._xe += tirelf_st._xedot * step;
            tirelf_st._ye += tirelf_st._yedot * step;
            tirelf_st._omega += tirelf_st._dOmega * step;
            // RF
            tirerf_st._xe += tirerf_st._xedot * step;
            tirerf_st._ye += tirerf_st._yedot * step;
            tirerf_st._omega += tirerf_st._dOmega * step;
            // LR
            tirelr_st._xe += tirelr_st._xedot * step;
            tirelr_st._ye += tirelr_st._yedot * step;
            tirelr_st._omega += tirelr_st._dOmega * step;
            // RR
            tirerr_st._xe += tirerr_st._xedot * step;
            tirerr_st._ye += tirerr_st._yedot * step;
            tirerr_st._omega += tirerr_st._dOmega * step;

            // Now the vehicle states
            if (veh_param._tcbool) {
                v_states._crankOmega += v_states._dOmega_crank * step;
            }

            // Integrate velocity level first
            v_states._u += v_states._udot * step;
            v_states._v += v_states._vdot * step;
            v_states._wx += v_states._wxdot * step;
            v_states._wz += v_states._wzdot * step;

            // Integrate position level next
            v_states._x += (v_states._u * cos(v_states._psi) - v_states._v * sin(v_states._psi)) * step;
            v_states._y += (v_states._u * sin(v_states._psi) + v_states._v * cos(v_states._psi)) * step;
            v_states._psi += v_states._wz * step;
            v_states._phi += v_states._wx * step;

            // Update time
            t += step;
            kernel_time += step;

            // Write to response if required -> regardless of no_outs or store_all we write all the vehicles to the
            // response
            if (output) {
                // The +1 here is because state at time 0 is not stored in device response
                if (abs(kernel_time - (timeStep_stored + 1) * dtout) < 1e-7) {
                    unsigned int time_offset = timeStep_stored * total_num_vehicles * collection_states;

                    device_response[time_offset + (total_num_vehicles * 0) + vehicle_id] = t;
                    device_response[time_offset + (total_num_vehicles * 1) + vehicle_id] = v_states._x;
                    device_response[time_offset + (total_num_vehicles * 2) + vehicle_id] = v_states._y;
                    device_response[time_offset + (total_num_vehicles * 3) + vehicle_id] = v_states._u;
                    device_response[time_offset + (total_num_vehicles * 4) + vehicle_id] = v_states._v;
                    device_response[time_offset + (total_num_vehicles * 5) + vehicle_id] = v_states._phi;
                    device_response[time_offset + (total_num_vehicles * 6) + vehicle_id] = v_states._psi;
                    device_response[time_offset + (total_num_vehicles * 7) + vehicle_id] = v_states._wx;
                    device_response[time_offset + (total_num_vehicles * 8) + vehicle_id] = v_states._wz;
                    device_response[time_offset + (total_num_vehicles * 9) + vehicle_id] = tirelf_st._omega;
                    device_response[time_offset + (total_num_vehicles * 10) + vehicle_id] = tirerf_st._omega;
                    device_response[time_offset + (total_num_vehicles * 11) + vehicle_id] = tirelr_st._omega;
                    device_response[time_offset + (total_num_vehicles * 12) + vehicle_id] = tirerr_st._omega;
                    timeStep_stored++;
                }
            }
        }
    }
}
// ======================================================================================================================
__global__ void Integrate(double current_time,
                          double kernel_sim_time,
                          double step,
                          bool output,
                          unsigned int total_num_vehicles,
                          unsigned int collection_states,
                          double dtout,
                          double* device_response,
                          SimDataNr* sim_data_nr,
                          SimStateNr* sim_states_nr) {
    double t = current_time;           // Set the current time
    double kernel_time = 0;            // Time since kernel was launched
    unsigned int timeStep_stored = 0;  // Number of time steps already stored in the device response

    unsigned int vehicle_id = blockIdx.x * blockDim.x + threadIdx.x;  // Get the vehicle id
    double end_time = (t + kernel_sim_time) - step / 10.;
    if (vehicle_id < total_num_vehicles) {
        while (t < end_time) {
            // Call the RHS to get accelerations for all the vehicles
            rhsFun(t, total_num_vehicles, sim_data_nr, sim_states_nr);
            // Extract the states of the vehicle and the tires
            VehicleState& v_states = sim_states_nr[vehicle_id]._veh_state;
            VehicleParam& veh_param = sim_data_nr[vehicle_id]._veh_param;
            TMeasyNrState& tirelf_st = sim_states_nr[vehicle_id]._tirelf_state;
            TMeasyNrState& tirerf_st = sim_states_nr[vehicle_id]._tirerf_state;
            TMeasyNrState& tirelr_st = sim_states_nr[vehicle_id]._tirelr_state;
            TMeasyNrState& tirerr_st = sim_states_nr[vehicle_id]._tirerr_state;

            // First the tire states
            // LF
            tirelf_st._omega += tirelf_st._dOmega * step;
            // RF
            tirerf_st._omega += tirerf_st._dOmega * step;
            // LR
            tirelr_st._omega += tirelr_st._dOmega * step;
            // RR
            tirerr_st._omega += tirerr_st._dOmega * step;

            // Now the vehicle states
            if (veh_param._tcbool) {
                v_states._crankOmega += v_states._dOmega_crank * step;
            }

            // Integrate velocity level first
            v_states._u += v_states._udot * step;
            v_states._v += v_states._vdot * step;
            v_states._wx += v_states._wxdot * step;
            v_states._wz += v_states._wzdot * step;
            // Integrate position level next
            v_states._x += (v_states._u * cos(v_states._psi) - v_states._v * sin(v_states._psi)) * step;
            v_states._y += (v_states._u * sin(v_states._psi) + v_states._v * cos(v_states._psi)) * step;
            v_states._psi += v_states._wz * step;
            v_states._phi += v_states._wx * step;

            // Update time
            t += step;
            kernel_time += step;

            // Write to response if required -> regardless of no_outs or store_all we write all the vehicles to the
            // response
            if (output) {
                // The +1 here is because state at time 0 is not stored in device response
                if (abs(kernel_time - (timeStep_stored + 1) * dtout) < 1e-7) {
                    unsigned int time_offset = timeStep_stored * total_num_vehicles * collection_states;

                    device_response[time_offset + (total_num_vehicles * 0) + vehicle_id] = t;
                    device_response[time_offset + (total_num_vehicles * 1) + vehicle_id] = v_states._x;
                    device_response[time_offset + (total_num_vehicles * 2) + vehicle_id] = v_states._y;
                    device_response[time_offset + (total_num_vehicles * 3) + vehicle_id] = v_states._u;
                    device_response[time_offset + (total_num_vehicles * 4) + vehicle_id] = v_states._v;
                    device_response[time_offset + (total_num_vehicles * 5) + vehicle_id] = v_states._phi;
                    device_response[time_offset + (total_num_vehicles * 6) + vehicle_id] = v_states._psi;
                    device_response[time_offset + (total_num_vehicles * 7) + vehicle_id] = v_states._wx;
                    device_response[time_offset + (total_num_vehicles * 8) + vehicle_id] = v_states._wz;
                    device_response[time_offset + (total_num_vehicles * 9) + vehicle_id] = tirelf_st._omega;
                    device_response[time_offset + (total_num_vehicles * 10) + vehicle_id] = tirerf_st._omega;
                    device_response[time_offset + (total_num_vehicles * 11) + vehicle_id] = tirelr_st._omega;
                    device_response[time_offset + (total_num_vehicles * 12) + vehicle_id] = tirerr_st._omega;
                    timeStep_stored++;
                }
            }
        }
    }
}
// ======================================================================================================================
__global__ void Integrate(double current_time,
                          double steering,
                          double throttle,
                          double braking,
                          double kernel_sim_time,
                          double step,
                          bool output,
                          unsigned int total_num_vehicles,
                          unsigned int collection_states,
                          double dtout,
                          double* device_response,
                          d18GPU::SimDataNr* sim_data_nr,
                          d18GPU::SimStateNr* sim_states_nr) {
    double t = current_time;           // Set the current time
    double kernel_time = 0;            // Time since kernel was launched
    unsigned int timeStep_stored = 0;  // Number of time steps already stored in the device response

    unsigned int vehicle_id = blockIdx.x * blockDim.x + threadIdx.x;  // Get the vehicle id
    double end_time = (t + kernel_sim_time) - step / 10.;
    if (vehicle_id < total_num_vehicles) {
        while (t < end_time) {
            // Call the RHS to get accelerations for all the vehicles
            rhsFun(t, total_num_vehicles, sim_data_nr, sim_states_nr, steering, throttle, braking);
            // Extract the states of the vehicle and the tires
            VehicleState& v_states = sim_states_nr[vehicle_id]._veh_state;
            VehicleParam& veh_param = sim_data_nr[vehicle_id]._veh_param;
            TMeasyNrState& tirelf_st = sim_states_nr[vehicle_id]._tirelf_state;
            TMeasyNrState& tirerf_st = sim_states_nr[vehicle_id]._tirerf_state;
            TMeasyNrState& tirelr_st = sim_states_nr[vehicle_id]._tirelr_state;
            TMeasyNrState& tirerr_st = sim_states_nr[vehicle_id]._tirerr_state;

            // First the tire states
            // LF
            tirelf_st._omega += tirelf_st._dOmega * step;
            // RF
            tirerf_st._omega += tirerf_st._dOmega * step;
            // LR
            tirelr_st._omega += tirelr_st._dOmega * step;
            // RR
            tirerr_st._omega += tirerr_st._dOmega * step;

            // Now the vehicle states
            if (veh_param._tcbool) {
                v_states._crankOmega += v_states._dOmega_crank * step;
            }

            // Integrate velocity level first
            v_states._u += v_states._udot * step;
            v_states._v += v_states._vdot * step;
            v_states._wx += v_states._wxdot * step;
            v_states._wz += v_states._wzdot * step;
            // Integrate position level next
            v_states._x += (v_states._u * cos(v_states._psi) - v_states._v * sin(v_states._psi)) * step;
            v_states._y += (v_states._u * sin(v_states._psi) + v_states._v * cos(v_states._psi)) * step;
            v_states._psi += v_states._wz * step;
            v_states._phi += v_states._wx * step;

            // Update time
            t += step;
            kernel_time += step;

            // Write to response if required -> regardless of no_outs or store_all we write all the vehicles to the
            // response
            if (output) {
                // The +1 here is because state at time 0 is not stored in device response
                if (abs(kernel_time - (timeStep_stored + 1) * dtout) < 1e-7) {
                    unsigned int time_offset = timeStep_stored * total_num_vehicles * collection_states;

                    device_response[time_offset + (total_num_vehicles * 0) + vehicle_id] = t;
                    device_response[time_offset + (total_num_vehicles * 1) + vehicle_id] = v_states._x;
                    device_response[time_offset + (total_num_vehicles * 2) + vehicle_id] = v_states._y;
                    device_response[time_offset + (total_num_vehicles * 3) + vehicle_id] = v_states._u;
                    device_response[time_offset + (total_num_vehicles * 4) + vehicle_id] = v_states._v;
                    device_response[time_offset + (total_num_vehicles * 5) + vehicle_id] = v_states._phi;
                    device_response[time_offset + (total_num_vehicles * 6) + vehicle_id] = v_states._psi;
                    device_response[time_offset + (total_num_vehicles * 7) + vehicle_id] = v_states._wx;
                    device_response[time_offset + (total_num_vehicles * 8) + vehicle_id] = v_states._wz;
                    device_response[time_offset + (total_num_vehicles * 9) + vehicle_id] = tirelf_st._omega;
                    device_response[time_offset + (total_num_vehicles * 10) + vehicle_id] = tirerf_st._omega;
                    device_response[time_offset + (total_num_vehicles * 11) + vehicle_id] = tirelr_st._omega;
                    device_response[time_offset + (total_num_vehicles * 12) + vehicle_id] = tirerr_st._omega;
                    timeStep_stored++;
                }
            }
        }
    }
}
