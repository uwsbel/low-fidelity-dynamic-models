// =============================================================================
// Authors: Huzaifa Unjhawala
// =============================================================================
//
// This demo describes simulating user provided number of HMMWVs (specified with JSON files), all operating on the 
// same driver inputs on the GPU. Since the Half Implicit solver is the only one supported for the GPU models,
// that is what is used here. The structure of the API is very similar to the CPU version except for the 
// additional requirements of specifying the number of vehicles and threads per block. 
// Use ./executable_name <total_number_of_vehicles> <threads_per_block>
//
// =============================================================================
#include <hip/hip_runtime.h>
#include <iostream>
#include <random>
#include <hip/hip_runtime.h>
#include <math.h>
#include <numeric>
#include <algorithm>
#include <iterator>
#include <chrono>

#include "dof11_halfImplicit_gpu.cuh"


using namespace d11GPU;
int main(int argc, char** argv) {
    // Get total number of vehicles from command line
    unsigned int num_vehicles = std::stoul(argv[1]);
    // Set the threads per block from command line
    unsigned int threads_per_block = std::stoul(argv[2]);

    std::string file_name = "double_lane4";
    // Driver inputs -> All vehicles have the same driver inputs
    std::string driver_file = "../data/input/" + file_name + ".txt";

    // Vehicle specification -> We assume that all vehicles have the same parameters
    std::string vehParamsJSON = (char*)"../data/json/HMMWV/vehicle.json";
    std::string tireParamsJSON = (char*)"../data/json/HMMWV/tmeasy.json";

    // Construct the solver
    d11SolverHalfImplicitGPU solver(num_vehicles);
    // The number of vehicles here sets these parameters and inputs for all these vehicles
    // If there is a need to set different parameters for different vehicles, then the solver
    // needs to be constructed for each vehicle separately (using the same solver object)
    solver.Construct(vehParamsJSON, tireParamsJSON, num_vehicles, driver_file);

    // Set the threads per block
    solver.SetThreadsPerBlock(threads_per_block);

    // Set the time step of the solver
    solver.SetTimeStep(1e-3);

    // Now we initialize the states -> These are all set to 0 (struct initializer)
    VehicleState veh_st;
    TMeasyState tiref_st;
    TMeasyState tirer_st;
    // Again we initialize the same states for all vehicles
    solver.Initialize(veh_st, tiref_st, tirer_st, num_vehicles);

    // Enable output for all the vehicles
    // solver.SetOutput("../data/output/" + file_name + "_hmmwv11", 100, true);
    // Enable output for 50 of the vehicles
    solver.SetOutput("../data/output/" + file_name + "_hmmwv11", 100, false, 50);
    // Set the simulation end time -> This is a input that *must* be set by the user
    solver.SetEndTime(22.0);

    // Solve
    // Time the solve
    auto start = std::chrono::high_resolution_clock::now();
    solver.Solve();
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed = end - start;
    std::cout << "Solve time: " << elapsed.count() << " s\n";
}
