// =============================================================================
// Authors: Huzaifa Unjhawala
// =============================================================================
//
// This demo describes simulating user provided number of HMMWVs (specified with JSON files) on a step-by-step
// basis. The "step" at which a new control input is provided can be set with SetKernelSimTime as shown here.
// Additionally, shown here is a way to provide a function object (functor) to the solver to provide the
// control inputs. This is done by creating a functor and passing it to the solver's SolveStep function.
// The functor needs to have an operator() function that takes in the current time and a pointer to the
// DriverInput struct. This functionality is useful when the control Functor represents some sort of controller.
// Since the Half Implicit solver is the only one supported for the GPU models, that is what is used here.
// When the solver is used in a step-by-step manner, the output is not stored in a file (unlike the CPU models).
// However, access to the vehicle states every control time step is provided through the GetSimState function.
// Use ./executable_name <total_number_of_vehicles> <threads_per_block>
//
// =============================================================================
#include <hip/hip_runtime.h>
#include <iostream>
#include <filesystem>
#include <random>
#include <hip/hip_runtime.h>
#include <math.h>
#include <numeric>
#include <algorithm>
#include <iterator>
#include <chrono>

#include "dof11_halfImplicit_gpu.cuh"

namespace fs = std::filesystem;
using namespace d11GPU;

// A simple functor for demonstration
struct MyFunctor {
    __device__ void operator()(double t, DriverInput* controls) {
        if (t < 1) {
            controls->m_steering = 0.0;
            controls->m_throttle = 0.0;
            controls->m_braking = 0.0;
        } else {
            controls->m_steering = 0.0;
            controls->m_throttle = 0.5;
            controls->m_braking = 0.0;
        }
    }
};

int main(int argc, char** argv) {
    if (argc != 3) {
        std::cerr << "Usage: " << argv[0] << " <total_number_of_vehicles> <threads_per_block>" << std::endl;
        return 1;
    }

    // Get total number of vehicles from command line
    unsigned int num_vehicles = std::stoul(argv[1]);
    unsigned int threads_per_block = std::stoul(argv[2]);

    std::string file_name = "acc3";
    std::string inputPath = "../../11dof-gpu/data/input/" + file_name + ".txt";

    // Vehicle specification -> We assume that all vehicles have the same parameters
    std::string vehParamsJSON = "../../11dof-gpu/data/json/HMMWV/vehicle.json";
    std::string tireParamsJSON = "../../11dof-gpu/data/json/HMMWV/tmeasy.json";

    // Ensure the input file exists
    if (!fs::exists(inputPath)) {
        std::cerr << "Error: Input file does not exist: " << inputPath << std::endl;
        return 1;
    }

    // Construct the solver
    d11SolverHalfImplicitGPU solver(num_vehicles);
    solver.Construct(vehParamsJSON, tireParamsJSON, num_vehicles);

    // Set the threads per block
    solver.SetThreadsPerBlock(threads_per_block);

    // Set the time step of the solver
    solver.SetTimeStep(1e-3);

    // Decide on the "step" timestep and set it here
    double control_time_step = 1e-1;
    solver.SetKernelSimTime(control_time_step);

    // Initialize the states
    VehicleState veh_st;
    TMeasyState tiref_st;
    TMeasyState tirer_st;
    solver.Initialize(veh_st, tiref_st, tirer_st, num_vehicles);

    // Initialize the controls functor
    MyFunctor myControlsFunctor;
    double endTime = 10.0;
    double timeStep = solver.GetStep();
    double t = 0;
    double new_time = 0;

    // Time the solve
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    while (t < (endTime - timeStep / 10.)) {
        new_time = solver.SolveStep(t, myControlsFunctor);  // Solve for the current time
        t = new_time;
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "Solve time (ms): " << milliseconds << "\n";

    // Extract terminal state of chosen vehicles and print the position
    SimState sim_state_1 = solver.GetSimState(0);
    std::cout << "X Position of vehicle 1: " << sim_state_1._veh_state._x << std::endl;
}
